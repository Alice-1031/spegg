#include "hip/hip_runtime.h"
#include "myConsumer.h"

myConsumer::myConsumer(int size_val, int maxsize_val, int seed_val, int ndemes, int species_ID_val) : coevolvingSpecie(size_val, maxsize_val, seed_val, ndemes, 0)
	{
	initialize_demes();

	for (int i=0; i < nloci; i++)
		{		
		draw_gaussian(size, 0.0, 1.0, fgenotype[i], gen);
		draw_gaussian(size, 0.0, 1.0, mgenotype[i], gen);

		// Just make the genotypes positive 
		for (int j=0; j < size; j++)
			{
			fgenotype[i][j] = fabs(fgenotype[i][j]);
			mgenotype[i][j] = fabs(mgenotype[i][j]);
			}
		}
	

	// To start, assign odd numbered individuals to be male, even numbered individuals to be female
	
	thrust::host_vector<int> twos(size);
	thrust::fill(twos.begin(), twos.end(), 2);
	thrust::transform(id.begin(), id.begin() + size, twos.begin(), sex.begin(), thrust::modulus<int>());
	
	//Set phenotype
	setPhenotype(0, size);	
	}


