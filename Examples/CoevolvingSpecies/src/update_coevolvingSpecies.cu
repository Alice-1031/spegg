#include "update_coevolvingSpecies.h"

void update_coevolvingSpecies::update()
	{
	for (int i=0; i < species[FOCAL_SPECIES_INDEX]->demeParameters->interacting_species.size(); i++)
		{
		prepare_interactions(i);
		interact();
		}
	}

// This nomenclature is confusing and should be fixed; target_species refers to the index AMONG interacting species, whilst ALTERNATIVE_SPECIES_INDEX refers to the index of target_species among all species.


void update_coevolvingSpecies::prepare_interactions(int target_species)
	{	
	ALTERNATIVE_SPECIES_INDEX = species[FOCAL_SPECIES_INDEX]->demeParameters->interacting_species[target_species];

	calculate_cumulative_deme_sizes();

	phenotype_used_in_interaction_focal_species = species[FOCAL_SPECIES_INDEX]->demeParameters->interaction_phenotype_indices[ALTERNATIVE_SPECIES_INDEX];
	phenotype_used_in_interaction_alternative_species = species[ALTERNATIVE_SPECIES_INDEX]->demeParameters->interaction_phenotype_indices[FOCAL_SPECIES_INDEX];

	// cast the vector pointers
	focal_species_phenotype = raw_pointer_cast(&species[FOCAL_SPECIES_INDEX]->phenotype[phenotype_used_in_interaction_focal_species][0]);
	alt_species_phenotype = raw_pointer_cast(&species[ALTERNATIVE_SPECIES_INDEX]->phenotype[phenotype_used_in_interaction_alternative_species][0]);
	alt_species_status = raw_pointer_cast(&species[ALTERNATIVE_SPECIES_INDEX]->status[0]);
	cumulative_deme_sizes_ptr = raw_pointer_cast(&cumulative_deme_sizes[0]);
	
	indices.resize(size);
	thrust::sequence(indices.begin(), indices.end());
	
	mean_number_of_others_sampled.resize(size);
	interaction_effects_on_fecundity.resize(size);
	interaction_effects_on_survivorship.resize(size);
	
	// Fan out the mean number of individuals that each individual of FOCAL_SPECIES will encounter
	thrust::gather(species[FOCAL_SPECIES_INDEX]->deme.begin(), species[FOCAL_SPECIES_INDEX]->deme.begin() + size, species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("mean_number_of_others_sampled"), mean_number_of_others_sampled.begin());

	// Fan out the interaction effects on fecundity by deme
	thrust::gather(species[FOCAL_SPECIES_INDEX]->deme.begin(), species[FOCAL_SPECIES_INDEX]->deme.begin() + size, species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_fecundity[target_species].begin(), interaction_effects_on_fecundity.begin());

	// Fan out the interaction effects on survivorship by deme
	thrust::gather(species[FOCAL_SPECIES_INDEX]->deme.begin(), species[FOCAL_SPECIES_INDEX]->deme.begin() + size, species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_survivorship[target_species].begin(), interaction_effects_on_survivorship.begin());

	// Unit testing code:
/* 	for (int i=0; i < size; i++)
		std::cout << " check that things were correctly setup for interaction between " << FOCAL_SPECIES_INDEX << " " << ALTERNATIVE_SPECIES_INDEX << " " << species[FOCAL_SPECIES_INDEX]->deme[i] << " " << interaction_effects_on_survivorship[i] << " " << interaction_effects_on_fecundity[i] <<  std::endl;
*/
	}

void update_coevolvingSpecies::interact()
	{
	interaction_kernel upit(FOCAL_SPECIES_INDEX, ALTERNATIVE_SPECIES_INDEX, focal_species_phenotype, alt_species_phenotype, alt_species_status, cumulative_deme_sizes_ptr);

	/* randomize the seeds */
	thrust::device_vector<unsigned int> seed_vals(size);
	unsigned int *seed_ptr = raw_pointer_cast(&seed_vals[0]);
	hiprandGenerate(species[FOCAL_SPECIES_INDEX]->gen, seed_ptr, size);
	
	/* determine the cost of the phenotype - i.e., the trade-off strength */
	thrust::device_vector<float> cost_of_interaction_phenotype(size);
	thrust::gather(species[FOCAL_SPECIES_INDEX]->deme.begin(), species[FOCAL_SPECIES_INDEX]->deme.begin() + size, species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("effect_of_interaction_phenotype_on_fecundity"), cost_of_interaction_phenotype.begin());

	/* perform the simulation */
	thrust::for_each(thrust::make_zip_iterator(
					 thrust::make_tuple(indices.begin(), 								    species[FOCAL_SPECIES_INDEX]->deme.begin(), 
							    mean_number_of_others_sampled.begin(),
							    species[FOCAL_SPECIES_INDEX]->status.begin(), 
							    species[FOCAL_SPECIES_INDEX]->phenotype[FECUNDITY_PHENOTYPE_INDEX].begin(),
							    species[FOCAL_SPECIES_INDEX]->phenotype[MORTALITY_PHENOTYPE_INDEX].begin(),
							    interaction_effects_on_fecundity.begin(), 
							    interaction_effects_on_survivorship.begin(),
							    cost_of_interaction_phenotype.begin(),
							    seed_vals.begin())),	
				 thrust::make_zip_iterator(
						thrust::make_tuple(indices.end(), 
							    species[FOCAL_SPECIES_INDEX]->deme.begin() + size,
							    mean_number_of_others_sampled.end(), 
							    species[FOCAL_SPECIES_INDEX]->status.begin() + size, 
							    species[FOCAL_SPECIES_INDEX]->phenotype[FECUNDITY_PHENOTYPE_INDEX].begin() + size,
							    species[FOCAL_SPECIES_INDEX]->phenotype[MORTALITY_PHENOTYPE_INDEX].begin() + size,									   
							    interaction_effects_on_fecundity.end(),
							    interaction_effects_on_survivorship.end(),
							    cost_of_interaction_phenotype.end(),
							    seed_vals.end())),
				 upit);

	hipDeviceSynchronize();
	}

void update_coevolvingSpecies::calculate_cumulative_deme_sizes()
	{
	cumulative_deme_sizes.resize(species[FOCAL_SPECIES_INDEX]->Num_Demes);
	thrust::inclusive_scan(species[ALTERNATIVE_SPECIES_INDEX]->deme_sizes.begin(), species[ALTERNATIVE_SPECIES_INDEX]->deme_sizes.begin() + species[FOCAL_SPECIES_INDEX]->Num_Demes, cumulative_deme_sizes.begin());
	}
