#include "hip/hip_runtime.h"
#include "update_coevolvingSpecies.h"
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>

void update_coevolvingSpecies::update()
	{
	for (int i=0; i < species[FOCAL_SPECIES_INDEX]->demeParameters->interacting_species.size(); i++)
		{
		prepare_interactions(i);
		interact(i);
		}
	}


// This nomenclature is confusing and should be fixed; target_species refers to the index AMONG interacting species, whilst ALTERNATIVE_SPECIES_INDEX refers to the index of target_species among all species.
void update_coevolvingSpecies::prepare_interactions(int target_species)
	{
	ALTERNATIVE_SPECIES_INDEX = species[FOCAL_SPECIES_INDEX]->demeParameters->interacting_species[target_species];

	calculate_cumulative_deme_sizes();

	phenotype_used_in_interaction_focal_species = species[FOCAL_SPECIES_INDEX]->demeParameters->interaction_phenotype_indices[ALTERNATIVE_SPECIES_INDEX];

	phenotype_used_in_interaction_alternative_species = species[ALTERNATIVE_SPECIES_INDEX]->demeParameters->interaction_phenotype_indices[FOCAL_SPECIES_INDEX];

	mean_number_of_others_sampled = *(species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("mean_number_of_others_sampled"));
	deme_specific_interaction_effects_on_fecundity = species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_fecundity[target_species][0];
	deme_specific_interaction_effects_on_survivorship = species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_survivorship[target_species][0];
	}

void update_coevolvingSpecies::calculate_cumulative_deme_sizes()
	{
	cumulative_deme_sizes.resize(species[FOCAL_SPECIES_INDEX]->Num_Demes);
	thrust::inclusive_scan(species[ALTERNATIVE_SPECIES_INDEX]->deme_sizes.begin(), species[ALTERNATIVE_SPECIES_INDEX]->deme_sizes.begin() + species[ALTERNATIVE_SPECIES_INDEX]->Num_Demes, cumulative_deme_sizes.begin());
	}

void update_coevolvingSpecies::interact(int target_species)
	{
	int deme_offset = 0;
	
	float fecundity_trait_trade_off = *(species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("effect_of_interaction_phenotype_on_fecundity"));
	float effect_of_interaction_on_fecundity = 1;
	float effect_of_interaction_on_survivorship = 1;

	int min_ind_index = 0;
	int max_ind_index = cumulative_deme_sizes[0];

	// Miscellaneous helper variables:
	int number_of_others_sampled, alternative_individual;
	float use_rv, weighted_effect_of_alternative_individual, old_fecundity, old_survivorship;

	for (int i=0; i < species[FOCAL_SPECIES_INDEX]->size; i++)
		{
		if (species[FOCAL_SPECIES_INDEX]->status[i] > 0)// If the individual is alive; this should be universally true; so no indentation for clarity.
		{
		// identify the deme_specific values; optimized to minimize memory access:
		if (species[FOCAL_SPECIES_INDEX]->deme[i] != deme_offset)
			{
			deme_offset = species[FOCAL_SPECIES_INDEX]->deme[i];
			mean_number_of_others_sampled = *(species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("mean_number_of_others_sampled") + deme_offset);

			deme_specific_interaction_effects_on_fecundity = species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_fecundity[target_species][deme_offset];
			deme_specific_interaction_effects_on_survivorship = species[FOCAL_SPECIES_INDEX]->demeParameters->deme_wise_interaction_effects_on_survivorship[target_species][deme_offset];

			fecundity_trait_trade_off = *(species[FOCAL_SPECIES_INDEX]->demeParameters->get_vector_ptr("effect_of_interaction_phenotype_on_fecundity") + deme_offset);
	
			min_ind_index = cumulative_deme_sizes[deme_offset - 1];
			max_ind_index = cumulative_deme_sizes[deme_offset];
			}

		float focal_individuals_phenotype = species[FOCAL_SPECIES_INDEX]->phenotype[phenotype_used_in_interaction_focal_species][i];

		if (min_ind_index != max_ind_index) // provided there is at least one other individual that is alive:
			{
			effect_of_interaction_on_fecundity = 1;
			effect_of_interaction_on_survivorship = 1;

			if (mean_number_of_others_sampled > MAX_SAMPLED)
				{
				// Use the discrete normal approximation to the poisson
				number_of_others_sampled = (int) round((double) mean_number_of_others_sampled + gsl_ran_gaussian(species[FOCAL_SPECIES_INDEX]->gen, sqrt(mean_number_of_others_sampled)));
				if (number_of_others_sampled < 0)
					number_of_others_sampled = 0;
				}
			else
				{
				number_of_others_sampled = gsl_ran_poisson(species[FOCAL_SPECIES_INDEX]->gen, mean_number_of_others_sampled);
				}

			for (int k=0; k < number_of_others_sampled; k++)
				{
				use_rv = gsl_rng_uniform(species[FOCAL_SPECIES_INDEX]->gen);
				alternative_individual = draw_ind(min_ind_index, max_ind_index, use_rv);
				weighted_effect_of_alternative_individual = 1;

				// Discount effect if the individual from the alternative species is dead, or if the individual sampled themselves. Note that alternative suitability criteria can be used here; e.g., if the alternative individual is a male, if the alternative individual is yourself, etc...
				if ( (species[ALTERNATIVE_SPECIES_INDEX]->status[alternative_individual] == 0) || ((FOCAL_SPECIES_INDEX == ALTERNATIVE_SPECIES_INDEX) && (alternative_individual == i) ) )
					{
					weighted_effect_of_alternative_individual = 0;	
					}

// For heterospecifics, the interaction strengths are mediated by trait values focal_phen1 and alt_phen1
				if ((FOCAL_SPECIES_INDEX != ALTERNATIVE_SPECIES_INDEX))
					{
					effect_of_interaction_on_fecundity = calculate_effect_of_interaction(effect_of_interaction_on_fecundity, weighted_effect_of_alternative_individual, deme_specific_interaction_effects_on_fecundity, focal_individuals_phenotype, species[ALTERNATIVE_SPECIES_INDEX]->phenotype[phenotype_used_in_interaction_alternative_species][alternative_individual]);
					effect_of_interaction_on_survivorship = calculate_effect_of_interaction(effect_of_interaction_on_survivorship, weighted_effect_of_alternative_individual, deme_specific_interaction_effects_on_survivorship, focal_individuals_phenotype, species[ALTERNATIVE_SPECIES_INDEX]->phenotype[phenotype_used_in_interaction_alternative_species][alternative_individual]);
					}
				else
					{
					effect_of_interaction_on_fecundity = calculate_effect_of_interaction_noPhen(effect_of_interaction_on_fecundity, weighted_effect_of_alternative_individual, deme_specific_interaction_effects_on_fecundity);
					effect_of_interaction_on_survivorship = calculate_effect_of_interaction_noPhen(effect_of_interaction_on_survivorship, weighted_effect_of_alternative_individual, deme_specific_interaction_effects_on_survivorship);
					}
				}
			
			}
		old_fecundity = species[FOCAL_SPECIES_INDEX]->phenotype[FECUNDITY_PHENOTYPE_INDEX][i];
		species[FOCAL_SPECIES_INDEX]->phenotype[FECUNDITY_PHENOTYPE_INDEX][i] = update_fecundity_phenotype(effect_of_interaction_on_fecundity, old_fecundity, focal_individuals_phenotype, fecundity_trait_trade_off);

		old_survivorship = species[FOCAL_SPECIES_INDEX]->phenotype[MORTALITY_PHENOTYPE_INDEX][i];
		species[FOCAL_SPECIES_INDEX]->phenotype[MORTALITY_PHENOTYPE_INDEX][i] = update_survivorship_phenotype(effect_of_interaction_on_survivorship, old_survivorship);
		}
		}
	}
