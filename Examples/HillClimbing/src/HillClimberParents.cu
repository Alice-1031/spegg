#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include "HillClimberParents.h"
#include "HillClimbers.h"

HillClimberParents::HillClimberParents(HillClimbers *species) : Parents(species)
	{
	FECUNDITY_PHENOTYPE_INDEX = demeParameters->species_specific_values["FECUNDITY_PHENOTYPE_INDEX"];
	}
	
void HillClimberParents::determine_probability_individual_becomes_female_parent()
	{
	int deme_offset = 0;
	float female_reproductive_skew = *(demeParameters->get_vector_ptr("FEMALE_REPRODUCTIVE_SKEW") + deme_offset);

	for (int i=0; i < size; i++)
		{
		// added to minimize memory access:
		if (deme[i] != deme_offset)
			{
			deme_offset = deme[i];
			female_reproductive_skew = *(demeParameters->get_vector_ptr("FEMALE_REPRODUCTIVE_SKEW") + deme_offset);
			}
		if (will_reproduceF[i] > 0)
			{
			probability_individual_becomes_female_parent[i] = powf(phenotype[FECUNDITY_PHENOTYPE_INDEX][i], female_reproductive_skew);
			}
		}
	}

void HillClimberParents::determine_probability_individual_becomes_male_parent()
	{
	int deme_offset = 0;
	float male_reproductive_skew = *(demeParameters->get_vector_ptr("MALE_REPRODUCTIVE_SKEW") + deme_offset);

	for (int i=0; i < size; i++)
		{
		// added to minimize memory access:
		if (deme[i] != deme_offset)
			{
			deme_offset = deme[i];
			male_reproductive_skew = *(demeParameters->get_vector_ptr("MALE_REPRODUCTIVE_SKEW") + deme_offset);
			}
		if (will_reproduceM[i] > 0)
			{
			probability_individual_becomes_male_parent[i] = powf(phenotype[FECUNDITY_PHENOTYPE_INDEX][i], male_reproductive_skew);
			}
		}
	}

