#include "hip/hip_runtime.h"
#include "HillClimbers_genotype_phenotype_maps.h"
#include <math/random_variables_functions.h>

#include <thrust/sequence.h>

#define MAX_MORTALITY 0.975
#define BASELINE_FECUNDITY 2

void fecundity_genotype_phenotype_map::calculate_phenotype(inds *species)
	{
	int deme_offset = 0;
	float coefficient_0 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF0") + deme_offset);
	float coefficient_1 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF1") + deme_offset);
	float coefficient_2 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF2") + deme_offset);
	float coefficient_3 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF3") + deme_offset);
	float coefficient_4 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF4") + deme_offset);

	for (int i=0; i < num_kids; i++)
		{
		if (deme_offset !=  species->deme[index_case + i])
			{
			deme_offset = species->deme[index_case + i];
			coefficient_0 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF0") + deme_offset);
			coefficient_1 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF1") + deme_offset);
			coefficient_2 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF2") + deme_offset);
			coefficient_3 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF3") + deme_offset);
			coefficient_4 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF4") + deme_offset);
			}
		species->phenotype[phenotype_index][index_case + i] = 0.5*coefficient_0*(species->fgenotype[0][index_case + i] + species->mgenotype[0][index_case + i]) 
								 + 0.5*coefficient_1*(species->fgenotype[1][index_case + i] + species->mgenotype[1][index_case + i]) 
								 + 0.5*coefficient_2*(species->fgenotype[2][index_case + i] + species->mgenotype[2][index_case + i]) 
								 + 0.5*coefficient_3*(species->fgenotype[3][index_case + i] + species->mgenotype[3][index_case + i]) 
								 + 0.5*coefficient_4*(species->fgenotype[4][index_case + i] + species->mgenotype[4][index_case + i]);

		species->phenotype[phenotype_index][index_case + i] = fabs(species->phenotype[phenotype_index][index_case + i]) + BASELINE_FECUNDITY;
		}
	}

void mortality_genotype_phenotype_map::calculate_phenotype(inds *species)
	{	
	int deme_offset = 0;
	float coefficient_0 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF0") + deme_offset);
	float coefficient_1 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF1") + deme_offset);
	float coefficient_2 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF2") + deme_offset);
	float coefficient_3 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF3") + deme_offset);
	float coefficient_4 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF4") + deme_offset);

	
	for (int i=0; i < num_kids; i++)
		{
		if (deme_offset !=  species->deme[index_case + i])
			{
			deme_offset = species->deme[index_case + i];
			coefficient_0 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF0") + deme_offset);
			coefficient_1 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF1") + deme_offset);
			coefficient_2 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF2") + deme_offset);
			coefficient_3 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF3") + deme_offset);
			coefficient_4 = *(Parameters->get_vector_ptr("GENPHEN_MAP_COEF4") + deme_offset);
			}
		species->phenotype[phenotype_index][index_case + i] = 0.5*coefficient_0*(species->fgenotype[5][index_case + i] + species->mgenotype[5][index_case + i]) 
								 + 0.5*coefficient_1*(species->fgenotype[6][index_case + i] + species->mgenotype[6][index_case + i]) 
								 + 0.5*coefficient_2*(species->fgenotype[7][index_case + i] + species->mgenotype[7][index_case + i]) 
								 + 0.5*coefficient_3*(species->fgenotype[8][index_case + i] + species->mgenotype[8][index_case + i]) 
								 + 0.5*coefficient_4*(species->fgenotype[9][index_case + i] + species->mgenotype[9][index_case + i]);

		// Make sure the resulting value is biologically meaningful:
		species->phenotype[phenotype_index][index_case + i] = exp(-1*fabs(species->phenotype[phenotype_index][index_case + i]));

		if (species->phenotype[phenotype_index][index_case + i] > MAX_MORTALITY)
			{
			species->phenotype[phenotype_index][index_case + i] = MAX_MORTALITY;
			}
		}
	}
 
