#include "hip/hip_runtime.h"
#include "update_Fish.h"
#include "Fish_Habitat.h"

// Update fecundity according to (reversible-irreversible*max_cond)/(max_cond*eggsize + egg_size)

void update_female_fecundity_functor(float &maximum_condition, float &irreversible_mass, float &reversible_mass, float &egg_size, float &fecundity)
	{
	fecundity = 0; // make sure there is no carry-over from the previous time step.
	fecundity = (reversible_mass - irreversible_mass*maximum_condition)/(maximum_condition*egg_size + egg_size);

	// because we are dealing with the number of kids, convert to integers
	int tempVal = (int) fecundity;
	fecundity = (float) tempVal;

	float surplus_mass_used_in_reproduction = (reversible_mass - irreversible_mass*maximum_condition);

	/* make sure that you deduct the amount used from reproductive females */
	if (fecundity > 0)
		{
		reversible_mass -= surplus_mass_used_in_reproduction;
		}
	else
		fecundity = 0 ;
	};

void update_Fish::update()
	{
	thrust::host_vector<float> resource_1_consumed(species->size);
	thrust::host_vector<float> resource_2_consumed(species->size);
	
	for (int i=0; i < species->Num_Demes; i++)
		{
		habitat->effect_of_inds_on_biotic_variable[0][i] = 0;
		habitat->effect_of_inds_on_biotic_variable[1][i] = 0;
		}
		
	thrust::host_vector<float> temp_eaten(Number_of_Demes);
	thrust::host_vector<int> temp_demes(Number_of_Demes);
	
	// Define variables:
	float xt0, yt0, xt1, yt1, kappa, maturity_threshold,  available_resources_1,  available_resources_2, common_satiation_resource_1, common_satiation_resource_2, satiation_resource_1, satiation_resource_2, prey_eaten_1, prey_eaten_2, maximum_consumption, percent_adult_resource_eaten, percent_juvenile_resource, Eg, rand;

	float functional_response_scalar_resource1, functional_response_scalar_resource2, handling_time_resource1, handling_time_resource2, F_sizes_at_maturity, M_sizes_at_maturity, mature_maximum_condition, juvenile_maximum_condition, conversion_efficiency, ons_constant, ons_coefficient, lw_coefficient, lw_exponent, effect_of_starvation, size_dependent_mortality_coefficient, size_dependent_mortality_constant, gamma_val, alpha_val;
	for (int Time_Step=0; Time_Step < intra_annual_time_steps; Time_Step++)
		{
		int offset = 0;
		for (int j=0; j < species->Num_Demes; j++)
			{
			if (j > 0)
				{
				offset += species->deme_sizes[j-1];
				}

			int individuals_deme = j;
			functional_response_scalar_resource1 = *(functional_response_scalar_resource1_ptr + individuals_deme);
			functional_response_scalar_resource2 = *(functional_response_scalar_resource2_ptr + individuals_deme) ;		
			handling_time_resource1= *(handling_time_resource1_ptr  + individuals_deme);
			handling_time_resource2 = *(handling_time_resource2_ptr  + individuals_deme); 

			F_sizes_at_maturity = *(F_sizes_at_maturity_ptr  + individuals_deme); 
			M_sizes_at_maturity = *(M_sizes_at_maturity_ptr  + individuals_deme); 
			mature_maximum_condition = *(mature_maximum_condition_ptr  + individuals_deme); 
			juvenile_maximum_condition = *(juvenile_maximum_condition_ptr  + individuals_deme); 
			conversion_efficiency = *(consumption_allometric_scalar_ptr  + individuals_deme); 

			ons_constant = *(ontogenetic_niche_shift_constant_ptr  + individuals_deme); 
			ons_coefficient = *(ontogenetic_niche_shift_coefficient_ptr  + individuals_deme); 
			lw_coefficient = *(length_weight_conversion_coefficient_ptr  + individuals_deme); 
			lw_exponent = *(length_weight_conversion_exponent_ptr  + individuals_deme);
			effect_of_starvation = *(effect_of_starvation_ptr  + individuals_deme); 
			size_dependent_mortality_coefficient = *(size_dependent_mortality_coefficient_ptr  + individuals_deme);
			size_dependent_mortality_constant = *(size_dependent_mortality_constant_ptr + individuals_deme) ;

			gamma_val = *(gamma_ptr + individuals_deme);
			alpha_val = *(alpha_g_ptr + individuals_deme);

			available_resources_1 = (habitat->biotic_variables[0][individuals_deme])/(habitat->prey_array[0]->prey_maximum_abundance[individuals_deme]);
			common_satiation_resource_1 = (functional_response_scalar_resource1*available_resources_1)/(1 + handling_time_resource1 * available_resources_1); 

			available_resources_2 = (habitat->biotic_variables[1][individuals_deme])/(habitat->prey_array[1]->prey_maximum_abundance[individuals_deme]);
			common_satiation_resource_2 = (functional_response_scalar_resource2*available_resources_2)/(1 + handling_time_resource2 * available_resources_2); 

			for (int i=offset; i < offset + species->deme_sizes[j]; i++)
				{
				if (species->status[i] > 0)
					{
					satiation_resource_1 = common_satiation_resource_1 + gsl_ran_gaussian(species->gen, 0);
					satiation_resource_2 = common_satiation_resource_2 + gsl_ran_gaussian(species->gen, 0);


					xt0 = species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i];
					yt0 = species->phenotype[REVERSIBLE_MASS_PHENOTYPE][i];

					xt1 = 0;
					yt1 = 0;

					kappa = 0;
			
					maturity_threshold = 0;

					if (species->sex[i] == 0)
						maturity_threshold = F_sizes_at_maturity;
					else
						maturity_threshold = M_sizes_at_maturity;

					// Set the initial kappa value. 
					// NB: Be very careful with this: "if (x) do_this if(!x) do_that" seems to give different instructions from "if (x) do_this else do_that"
					if (xt0 >= maturity_threshold)
						kappa = 1/((1+mature_maximum_condition)*mature_maximum_condition);
					else
						kappa = 1/((1+juvenile_maximum_condition)*juvenile_maximum_condition);

					//printf("kappa: %f\n", kappa);

					// Do the dynamic energy budgeting

					maximum_consumption = 0;
					percent_adult_resource_eaten = 0;
					percent_juvenile_resource = 0;
					Eg = 0;
					prey_eaten_1 = 0;
					prey_eaten_2 = 0;

					maximum_consumption = conversion_efficiency *pow(xt0+yt0, gamma_val);

					//printf("Maximum consumption: %f\n", maximum_consumption);
					// Energy growth is therefore
					percent_adult_resource_eaten = 1/(1+exp(-(ons_constant + ons_coefficient*(lw_coefficient* pow(xt0,lw_exponent)) + gsl_ran_gaussian(species->gen, 0.0))));
					percent_juvenile_resource = 1-percent_adult_resource_eaten;

					prey_eaten_1 += satiation_resource_1*percent_juvenile_resource*maximum_consumption;
					prey_eaten_2 += satiation_resource_2*percent_adult_resource_eaten*maximum_consumption;

					//printf("Prey eaten juv. v. adult: %f %f ONS juv v. ad.: %f %f Size: %f\n", prey_eaten_1, prey_eaten_2, percent_juvenile_resource, percent_adult_resource_eaten, xt0);

					// Subtract metabolic costs
					Eg = (satiation_resource_1*percent_juvenile_resource + satiation_resource_2*percent_adult_resource_eaten)*maximum_consumption - alpha_val*(xt0+yt0);

					if (Eg >= 0)
						{
						xt1 = (yt0/xt0)*kappa*Eg + xt0;
						yt1 = (1-(yt0/xt0)*kappa)*Eg + yt0;
						}
					else
						{
						yt1 = yt0 + Eg;
						if (yt1 < 0)
							{
							yt1 = 0;
							}
						xt1=xt0;		
						}
					//printf("Juvenile Prey available: %f Energy gained: %f Original size: %f %f Final size: %f %f\n", available_resources_1, Eg, xt0, yt0, xt1, yt1);

					species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i] = xt1;
					species->phenotype[REVERSIBLE_MASS_PHENOTYPE][i] = yt1;
				
					// Update the resources eaten	
					habitat->effect_of_inds_on_biotic_variable[0][individuals_deme] += prey_eaten_1;
					habitat->effect_of_inds_on_biotic_variable[1][individuals_deme] += prey_eaten_2;


	/**********************
	Update vital rates
	**********************/
					if (Time_Step == intra_annual_time_steps - 1)
						{
						// Update fecundity for females
						if (species->sex[i] == 0)
							{
							update_female_fecundity_functor(juvenile_maximum_condition, species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i], species->phenotype[REVERSIBLE_MASS_PHENOTYPE][i], species->phenotype[EGGSIZE_PHENOTYPE][i], species->phenotype[FECUNDITY_PHENOTYPE][i]);			
							}
						}
					// Determine survivorship:
					float condition = species->phenotype[REVERSIBLE_MASS_PHENOTYPE][i] / species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i];
	
					species->phenotype[MORTALITY_PHENOTYPE][i] = (1-exp(-effect_of_starvation*condition))* (1/(1+exp(size_dependent_mortality_coefficient*(species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i] + size_dependent_mortality_constant))));

					if (species->phenotype[MORTALITY_PHENOTYPE][i] > 1)
						{
						species->phenotype[MORTALITY_PHENOTYPE][i] = 1;
						}
					if (species->phenotype[MORTALITY_PHENOTYPE][i] < 0)
						{
						species->phenotype[MORTALITY_PHENOTYPE][i] = 0;
						}

					// Leads to a maximum survivorship of about 30 years
					species->phenotype[MORTALITY_PHENOTYPE][i] = 0.9997*species->phenotype[MORTALITY_PHENOTYPE][i];
	

					rand = gsl_rng_uniform(species->gen);
				
					//printf("%f\n", species->phenotype[MORTALITY_PHENOTYPE][i] );
					//printf("Condition: %f Body size: %f Reversible mass: %f Daily Surivovrship: %f\n", condition, species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE][i], species->phenotype[REVERSIBLE_MASS_PHENOTYPE][i], species->phenotype[MORTALITY_PHENOTYPE][i] );
					if (rand < species->phenotype[MORTALITY_PHENOTYPE][i])
						{
						species->status[i] = 1;
						}
					else
						{
						species->status[i] = 0;
						}
					}
				}
			}

		habitat->update();
		}
	}

void update_Fish::prepare_survivorship_constants_pointers()
	{
	effect_of_starvation_ptr = (species->demeParameters->get_vector_ptr("effect_of_starvation"));
	size_dependent_mortality_constant_ptr = (species->demeParameters->get_vector_ptr("size_dependent_mortality_constant"));
	size_dependent_mortality_coefficient_ptr = (species->demeParameters->get_vector_ptr("size_dependent_mortality_coefficient"));
	}

void update_Fish::prepare_growth_constants_pointers()
	{
	consumption_allometric_scalar_ptr = (species->demeParameters->get_vector_ptr("conversion_efficiency"));
	
	gamma_ptr = (species->demeParameters->get_vector_ptr("gamma_exponent"));
	ontogenetic_niche_shift_constant_ptr = (species->demeParameters->get_vector_ptr("ontogenetic_niche_shift_constant"));
	ontogenetic_niche_shift_coefficient_ptr = (species->demeParameters->get_vector_ptr("ontogenetic_niche_shift_coefficient"));
	length_weight_conversion_coefficient_ptr = (species->demeParameters->get_vector_ptr("length_weight_conversion_coefficient"));
	length_weight_conversion_exponent_ptr = (species->demeParameters->get_vector_ptr("length_weight_conversion_exponent"));

	handling_time_resource1_ptr = (species->demeParameters->get_vector_ptr("handling_time"));
	handling_time_resource2_ptr = (species->demeParameters->get_vector_ptr("handling_time"));

	functional_response_scalar_resource1_ptr = (species->demeParameters->get_vector_ptr("functional_response_numerator"));
	functional_response_scalar_resource2_ptr = (species->demeParameters->get_vector_ptr("functional_response_numerator"));

	/* constants governing growth rates */
	alpha_g_ptr = (species->demeParameters->get_vector_ptr("alpha_g"));

	mature_maximum_condition_ptr = (species->demeParameters->get_vector_ptr("mature_maximum_condition"));
	juvenile_maximum_condition_ptr = (species->demeParameters->get_vector_ptr("juvenile_maximum_condition"));

	M_sizes_at_maturity_ptr = (species->demeParameters->get_vector_ptr("M_sizes_at_maturity"));
	F_sizes_at_maturity_ptr = (species->demeParameters->get_vector_ptr("F_sizes_at_maturity"));

	effect_of_starvation_ptr = (species->demeParameters->get_vector_ptr("effect_of_starvation"));
	size_dependent_mortality_constant_ptr = (species->demeParameters->get_vector_ptr("size_dependent_mortality_constant"));
	size_dependent_mortality_coefficient_ptr = (species->demeParameters->get_vector_ptr("size_dependent_mortality_coefficient"));
	}

