#include <math/thrust_prob_table.h>

#include <thrust/binary_search.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

void ThrustProbTable::setup(thrust::host_vector<float>::iterator prob_begin, thrust::host_vector<float>::iterator prob_end)
	{
	int n = thrust::distance(prob_begin, prob_end);
	cumulative_prob.resize(n);
	
	float total = thrust::reduce(prob_begin, prob_end);
	thrust::host_vector<float> total_vec(n);
	thrust::fill(total_vec.begin(), total_vec.end(), total);
	
	thrust::host_vector<float> relative_prob(n);
	thrust::transform(prob_begin, prob_end, total_vec.begin(), relative_prob.begin(), thrust::divides<float>());
	
	thrust::inclusive_scan(relative_prob.begin(), relative_prob.end(), cumulative_prob.begin());
	}

void ThrustProbTable::draw(thrust::host_vector<float>::iterator uniform_begin, thrust::host_vector<float>::iterator uniform_end, thrust::host_vector<int>::iterator result)
	{
	thrust::lower_bound(cumulative_prob.begin(), cumulative_prob.end(), uniform_begin, uniform_end, result);
	}
