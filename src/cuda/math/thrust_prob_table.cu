#include <math/thrust_prob_table.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

void ThrustProbTable::setup(thrust::device_vector<float>::iterator prob_begin, thrust::device_vector<float>::iterator prob_end)
	{
	int n = thrust::distance(prob_begin, prob_end);
	cumulative_prob.resize(n);
	
	float total = thrust::reduce(prob_begin, prob_end);
	thrust::device_vector<float> total_vec(n);
	thrust::fill(total_vec.begin(), total_vec.end(), total);
	
	thrust::device_vector<float> relative_prob(n);
	thrust::transform(prob_begin, prob_end, total_vec.begin(), relative_prob.begin(), thrust::divides<float>());
	
	thrust::inclusive_scan(relative_prob.begin(), relative_prob.end(), cumulative_prob.begin());
	}

void ThrustProbTable::draw(thrust::device_vector<float>::iterator uniform_begin, thrust::device_vector<float>::iterator uniform_end, thrust::device_vector<int>::iterator result)
	{
	thrust::lower_bound(cumulative_prob.begin(), cumulative_prob.end(), uniform_begin, uniform_end, result);
	}

thrust::device_ptr<float> ThrustProbTable::get_cumulative_probabilities_ptr()
	{
	return(&cumulative_prob[0]);
	}
