#include "sampling_event.h"
#include "Sample_With_Replacement.h"
#include "Sample_without_Replacement_1Pass.h"
// #include "Sample_without_Replacement.h" // For now, this isn't quite ready yet

// TODO: right now have no way of returning NULL if error?

SamplingEvent *SamplingEvent::create_SamplingEvent(SamplingInput *sampling_input, hiprandGenerator_t gen)
	{
	if (sampling_input->sampling_scheme == 1)
		return new Sample_With_Replacement(sampling_input, gen);

	if (sampling_input->sampling_scheme == 2)
		return new Sample_without_Replacement_1Pass(sampling_input, gen);

 /*
// not ready 
  if (sampling_input->sampling_scheme == 3)
    return new Sample_without_Replacement(sampling_input, gen);

*/
	}


