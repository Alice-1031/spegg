#include <species/add_kids/neonates_class.h>

/* One problem with a separate neonates class is that all this stuff has to get copied back into example species. It might just not be worth it depending on the performance cost. */

EggsNeonates::EggsNeonates(inds_stochastic *species, thrust::host_vector<int> &kids_per_mom) 
	{
	this->species = species;
	this->gen = species->gen;

	nphen = species->nphen;
	nloci = species->nloci;

	//Copy in the recombination map
	recomb_rate.resize(nloci);
	thrust::copy(species->demeParameters->GeneticArchitecture->recombination_rates.begin(), species->demeParameters->GeneticArchitecture->recombination_rates.begin() + nloci, recomb_rate.begin());

	previous_pop_size = species->size;

	Num_Demes = species->Num_Demes;
	Neonates_per_Deme.resize(Num_Demes);
	thrust::fill(Neonates_per_Deme.begin(), Neonates_per_Deme.end(), 0);

	Determine_Neonate_Population_Sizes(species->demeParameters, species->deme, kids_per_mom, species->deme_sizes, species->max_deme_sizes);

	kids_deme.resize(Total_Number_of_Neonates);

	amplify_sequence( Neonates_per_Deme, Num_Demes, kids_deme );

	integrate_kids();
	mutation_magnitude.resize(Total_Number_of_Neonates);
	mutation_rate.resize(Total_Number_of_Neonates);
	}

void EggsNeonates::Determine_Neonate_Population_Sizes(DemeSettings *demeParameters,
						      thrust::host_vector<int> &everybodys_deme,
						      thrust::host_vector<int> &kids_per_mom,
						      thrust::host_vector<int> &current_deme_sizes,
						      thrust::host_vector<int> &maximum_deme_sizes)
	{
	/*
	* Because there are often more neonates than max_deme_sizes, this function culls the surplus neonates at random by determining the number of neonates each deme can contribute
	*/
	hipDeviceSynchronize();
	reduce_by_key_with_zeros(everybodys_deme, kids_per_mom, Neonates_per_Deme, previous_pop_size, Num_Demes); 

	// Make sure no population has more kids than there are spaces
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(Neonates_per_Deme.begin(), current_deme_sizes.begin(), maximum_deme_sizes.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(Neonates_per_Deme.end(), current_deme_sizes.end(), maximum_deme_sizes.end())),
	                 adjust_kids_functor());

	Total_Number_of_Neonates = thrust::reduce(Neonates_per_Deme.begin(), Neonates_per_Deme.end());
	}


void EggsNeonates::inherit_genotypes(thrust::host_vector<float> &probability_individuals_become_mothers,
				 thrust::host_vector<float> &probability_individuals_become_fathers)
	{
	get_maternally_derived_genotype(probability_individuals_become_mothers, species->mgenotype, species->fgenotype);
	get_paternally_derived_genotype(probability_individuals_become_fathers, species->mgenotype, species->fgenotype);
	mutate(species->fgenotype, species->mgenotype);
	}


void EggsNeonates::get_maternally_derived_genotype(thrust::host_vector<float> &probability_individuals_become_mothers,
					     thrust::host_vector<float> *&mgenotype,
					     thrust::host_vector<float> *&fgenotype)
	{
	/*
	* Calculate the offspring's genotypes at the maternally inherited loci. This function will determine who the mother is, and generate a haploid gamete from the mother that will be grafted on to the offspring.
	*/
	mating_ThrustProbTable_demes at;
	thrust::host_vector<int> mother_index(Total_Number_of_Neonates);
	thrust::host_vector<float> rand(Total_Number_of_Neonates);
	thrust::host_vector<int> parity(Total_Number_of_Neonates);

/*
	Feed reproductive probablity into the setup of the alias table.
	Draw from the alias table to determine mothers.
*/
	at.setup(probability_individuals_become_mothers.begin(), probability_individuals_become_mothers.begin() + previous_pop_size);
	for (int i=0; i < rand.size(); i++)
		{
		rand[i] = gsl_rng_uniform(gen);
		}	
	at.determine_key_offsets( Num_Demes, species->deme_sizes );
	at.adjust_randoms(rand.begin(), rand.end(), kids_deme.begin(), kids_deme.end());

	at.draw(rand.begin(), rand.end(), mother_index.begin());
	thrust::copy(mother_index.begin(), mother_index.end(), species->maternal_id.begin() + previous_pop_size);

//Initialize parity to zeroes
//Parity vector is used to keep track of where the recombination is happening.
	thrust::fill(parity.begin(), parity.end(), 0);


//Recombination for fgenotype
	for (int i = 0 ; i < nloci ; i++) 
		{
		for (int j=0; j < rand.size(); j++)
			{
			rand[j] = gsl_rng_uniform(gen);
			}
		recombine(rand, mother_index, parity, fgenotype, mgenotype, fgenotype[i], i);
		}
	}

void EggsNeonates::get_paternally_derived_genotype(thrust::host_vector<float> &probability_individuals_become_fathers,
					     thrust::host_vector<float> *&mgenotype,
					     thrust::host_vector<float> *&fgenotype)
	{
	/*
	* Calculate the offspring's genotypes at the paternally inherited loci. This function will determine who the mother is, and generate a haploid gamete from the mother that will be grafted on to the offspring.
	*/
	mating_ThrustProbTable_demes at;
	thrust::host_vector<int> father_index(Total_Number_of_Neonates);
	thrust::host_vector<float> rand(Total_Number_of_Neonates);
	thrust::host_vector<int> parity(Total_Number_of_Neonates);
	
	/*
	Directly use phenotype[1] as the reproductive probability.
	Reproductive probability = phenotype[1].
	Feed reproductive probablity into the setup of the alias table.
	Draw from the alias table to determine fathers.
	*/
	at.setup(probability_individuals_become_fathers.begin(), probability_individuals_become_fathers.begin() + previous_pop_size);

	for (int i=0; i < rand.size(); i++)
		{
		rand[i] = gsl_rng_uniform(gen);
		}

	at.determine_key_offsets( Num_Demes, species->deme_sizes );	
	at.adjust_randoms(rand.begin(), rand.end(), kids_deme.begin(), kids_deme.end());

	at.draw(rand.begin(), rand.end(), father_index.begin());
	thrust::copy(father_index.begin(), father_index.end(), species->paternal_id.begin() + previous_pop_size);


	//Reset parity to zeroes
	thrust::fill(parity.begin(), parity.end(), 0);

	//Recombination for mgenotype
	for (int i = 0 ; i < nloci ; i++) 
		{
		for (int j=0; j < rand.size(); j++)
			{
			rand[j] = gsl_rng_uniform(gen);
			}
		recombine(rand, father_index, parity, fgenotype, mgenotype, mgenotype[i], i);
		}
	}

void EggsNeonates::mutate(thrust::host_vector<float> *&mgenotype, thrust::host_vector<float> *&fgenotype)
	{
	/*
	* Determine, for each offspring's locus, whether there will be a mutation at that locus, and if so what the magnitude of that mutation will be and how that changes the offspring's allelic value. The current behavior assumes mutations are gaussian about the parental allelic value, and that the mutation parameters (mutation rate and sd of gaussian) vary by deme. Possible expansions include allowing the mutation rate itself to be an individual-specific phenotype, or alternative mutational models (e.g., point mutations that have a categorical rather than quantitative effect.
	*/
	thrust::host_vector<float> mutation_size(2*Total_Number_of_Neonates); 

	thrust::copy(mutation_magnitude.begin(), mutation_magnitude.begin() + Total_Number_of_Neonates, mutation_size.begin());
	thrust::copy(mutation_magnitude.begin(), mutation_magnitude.begin() + Total_Number_of_Neonates, mutation_size.begin() + Total_Number_of_Neonates);

	thrust::host_vector<float> mutation(2*Total_Number_of_Neonates);

	// no mutation at the original sex determining locus
	for (int i = 1 ; i < nloci ; i++) 
		{
		thrust::gather(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->demeParameters->GeneticArchitecture->get_mutation_magnitudes_ptr(i), mutation_magnitude.begin());
		thrust::gather(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->demeParameters->GeneticArchitecture->get_mutation_rates_ptr(i), mutation_rate.begin());
		
		for (int j=0; j < Total_Number_of_Neonates; j++)
			{
			int mutation_event = gsl_ran_bernoulli(gen, mutation_rate[j]); 
			if (mutation_event == 1)
				{
				mgenotype[i][j] = mgenotype[i][j] + gsl_ran_gaussian(gen, mutation_magnitude[j]);
				}
		
			mutation_event = gsl_ran_bernoulli(gen, mutation_magnitude[j]);

			if (mutation_event == 1)
				{
				fgenotype[i][j] = fgenotype[i][j] + gsl_ran_gaussian(gen, mutation_magnitude[j]);
				}
			}
		}
	}


void EggsNeonates::recombine(thrust::host_vector<float> &rand,
			     thrust::host_vector<int> &parent,
			     thrust::host_vector<int> &parity,
			     thrust::host_vector<float> *&parents_fgenotype,
			     thrust::host_vector<float> *&parents_mgenotype,
			     thrust::host_vector<float> &kids_genotype,
			     int locus_ID)
	{
/*
* A method that implements recombination. Note that in many cases, the arguments for either (parents_fgenotype or parents_mgenotype) and kids_genotype will be the same vector. This function increments the kids_genotype to begin at the (previous_pop_size)th index. This should probably be refactored so that there will be a kids_mgenotype and kids_fgenotype vector that is local to the neonates class, and that later gets copied into the inds class, i.e. something like: 
\code{.cpp}
	thrust::copy(kids_fgenotype[i].begin(), kids_fgenotype[i].end(), fgenotype[i].begin() + size);
\endcode
*/
	//Set up recombination functor.
	float *fgenotype_ptr = &parents_fgenotype[locus_ID][0];
	float *mgenotype_ptr = &parents_mgenotype[locus_ID][0];
	recombination_functor rfunc(fgenotype_ptr, mgenotype_ptr, recomb_rate[locus_ID]);
	
	//Perform recombination with arbitrary transform.
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(parent.begin(), rand.begin(), parity.begin(), kids_genotype.begin() + previous_pop_size)),
			 thrust::make_zip_iterator(thrust::make_tuple(parent.begin() + Total_Number_of_Neonates, rand.begin() + Total_Number_of_Neonates, parity.begin() + Total_Number_of_Neonates, kids_genotype.begin() +  previous_pop_size + Total_Number_of_Neonates)),
			 rfunc);
	}


void EggsNeonates::integrate_kids()
	{
	thrust::copy(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->deme.begin() + previous_pop_size);
	thrust::sequence(species->id.begin() + previous_pop_size, species->id.begin() + previous_pop_size + Total_Number_of_Neonates, species->nextid);
	thrust::fill(species->status.begin() + previous_pop_size, species->status.begin() + previous_pop_size + Total_Number_of_Neonates, 1);
	thrust::fill(species->age.begin() + previous_pop_size, species->age.begin() + previous_pop_size + Total_Number_of_Neonates, 0);
	species->nextid += Total_Number_of_Neonates;
	species->size += Total_Number_of_Neonates;
	}
