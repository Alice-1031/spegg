#include "environment.h"

#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

environment::environment(int seed_val, int num_biotic_variables, int num_abiotic_variables, int num_demes) : seed(seed_val), ndemes(num_demes), nbiotic_vars(num_biotic_variables), nabiotic_vars(num_abiotic_variables)
	{
	//Allocate biotic data vectors.
	biotic_variables = new thrust::device_vector<float>[nbiotic_vars];
	effect_of_inds_on_biotic_variable = new thrust::device_vector<float>[nbiotic_vars];

	// Resize to accord with number of demes
	for (int i = 0 ; i < nbiotic_vars ; i++) 
		{
		biotic_variables[i].resize(ndemes);
		effect_of_inds_on_biotic_variable[i].resize(ndemes);
		}

		// Initialize the first part of the feedback process 
	for (int i = 0 ; i < nbiotic_vars ; i++) 
		{
		thrust::fill(effect_of_inds_on_biotic_variable[i].begin(), effect_of_inds_on_biotic_variable[i].begin() + ndemes, 0);
		}

	// Allocate abiotic data vectors
	abiotic_variables = new thrust::device_vector<float>[nabiotic_vars];
	
	for (int i=0; i < nabiotic_vars; i++)
		{
		abiotic_variables[i].resize(ndemes);
		}

	//Initialize hiprand generator.
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	}

thrust::device_ptr<float> environment::get_abiotic_vector_ptr(const char *abiotic_variable_name)
	{
	return(&abiotic_variables[abiotic_variable_indices[abiotic_variable_name]][0]);
	}


environment::~environment()
	{
	delete[] biotic_variables;
	delete[] effect_of_inds_on_biotic_variable;
	}


